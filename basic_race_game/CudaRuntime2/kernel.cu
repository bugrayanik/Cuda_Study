#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <string.h>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <chrono>
#include <thread>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
#include "Racers.h"
#include "Racer.h"

//note that to get use of atomicAdd function, you need to use compute_20.sm_21 compilation flag

using namespace std;


class Game
{
public:
    int  data_size;   // Set from host
    __host__ Game();
    __host__ ~Game();
    __host__ void setValues(int size);
    __device__ void dosomething(int, int*);
    __host__ int* export_data();
    __host__ void free_data();
    __host__ void prepareDeviceObj();
    __host__ void retrieveDataToHost();
    

private:
    int* data; // device side
    int* h_data; //host side
};

__host__ Game::Game()
{
}

__host__ Game::~Game()
{
}

__host__ void Game::prepareDeviceObj() {
    hipMemcpy(data, h_data, data_size * sizeof(h_data[0]), hipMemcpyHostToDevice);
}
__host__ void Game::retrieveDataToHost() {
    hipMemcpy(h_data, data, data_size * sizeof(h_data[0]), hipMemcpyDeviceToHost);
    
}

__host__ void Game::setValues( int size)
{
    data_size = size;
    hipMalloc(&data, data_size * sizeof(data[0]));
    h_data = (int*)malloc(data_size * sizeof(h_data[0]));
    memset(h_data, 0, data_size * sizeof(h_data[0]));
}

__device__ void Game::dosomething(int idx, int* newspeed)

{
    int toAdd = newspeed[idx];
    atomicAdd(&data[idx], toAdd);
    
}

__host__ void Game::free_data() {

    hipFree(data);
    free(h_data);
}

__host__ int* Game::export_data() {

    return h_data;
}


__global__ void myKernel(Game obj, int* h_result)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < obj.data_size)
    {
        obj.dosomething(idx,h_result);
       
    }
}
__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
    int id = threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}
__global__ void generate(hiprandState* globalState, int* result, int* max, int* min, int count)
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform(&localState);
    globalState[ind] = localState;
    if (ind < count)
        result[ind] = truncf(*min + (*max - *min) * RANDOM);
}

Racers Real_Tabela;
Game globalInstance;

int main(int argc, char** argv){

   
    for(int i=1;i<101;i++)
    Real_Tabela.add_racer(0,0,i);
    Real_Tabela.display_tabela();

    int N = 100; // no of random numbers to be generated
    int MIN = 1; // max range of random number
    int MAX = 5; // min range of random number
    dim3 tpb(N, 1, 1);

    
    globalInstance.setValues(100);
    

    for (int i = 1; i < 101; i++ ) {
        
        hiprandState* devStates;
        hipMalloc(&devStates, N * sizeof(hiprandState));
        // setup seeds 
        setup_kernel << < 1, tpb >> > (devStates, time(NULL));
        hipDeviceSynchronize();
        int* d_result, * h_result;
        hipMalloc(&d_result, N * sizeof(int));
        h_result = (int*)malloc(N * sizeof(int));
        int* d_max, * h_max, * d_min, * h_min;
        hipMalloc(&d_max, sizeof(int));
        h_max = (int*)malloc(sizeof(int));
        hipMalloc(&d_min, sizeof(int));
        h_min = (int*)malloc(sizeof(int));
        *h_max = MAX;
        *h_min = MIN;
        hipMemcpy(d_max, h_max, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_min, h_min, sizeof(int), hipMemcpyHostToDevice);
        // generate random numbers
        generate << < 1, tpb >> > (devStates, d_result, d_max, d_min, N);
        hipDeviceSynchronize();

        hipMemcpy(h_result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);
        globalInstance.prepareDeviceObj();
        myKernel <<<1, 100 >>> (globalInstance, d_result);
        hipDeviceSynchronize();
        globalInstance.retrieveDataToHost();
        
        int* position = globalInstance.export_data();
        Real_Tabela.modify_variable_of(position,h_result);
        Real_Tabela.display_tabela();

        if (Real_Tabela.check_winner() == true)
            break;

        this_thread::sleep_for(chrono::milliseconds(500));
        hipDeviceReset();
    };
    globalInstance.free_data();

    Real_Tabela.display_winner();

    return 0;
}